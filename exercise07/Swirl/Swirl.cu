#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include "common.h"
#include "bmp.h"
#include <stdlib.h>
#include <GLUT/glut.h>

#define DIM 512
#define blockSize 8

#define PI 3.1415926535897932f
#define centerX (DIM/2)
#define centerY (DIM/2)

float sourceColors[DIM*DIM];	// host memory for source image
float readBackPixels[DIM*DIM];	// host memory for swirled image

float *sourceDevPtr;			// device memory for source image
float *swirlDevPtr;				// device memory for swirled image

__global__ void swirlKernel( float *sourcePtr, float *targetPtr ) 
{
	int index = threadIdx.x;    
	// TODO: Index berechnen	

	// TODO: Den swirl invertieren.

	targetPtr[index] = sourcePtr[index];    // simple copy
}

void display(void)	
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	dim3    blocks(DIM/blockSize,DIM/blockSize);
	dim3    threads(blockSize,blockSize);	

	//: Swirl Kernel aufrufen.
	swirlKernel<<<blocks, threads, 0>>>(sourceDevPtr,swirlDevPtr);
	// Ergebnis zu host memory zuruecklesen.
	CUDA_SAFE_CALL( hipMemcpy(readBackPixels, swirlDevPtr, DIM * DIM, hipMemcpyDeviceToHost) );	

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels( DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels );

	glutSwapBuffers();
}

// clean up memory allocated on the GPU
void cleanup() {
    CUDA_SAFE_CALL( hipFree( sourceDevPtr ) ); 
    CUDA_SAFE_CALL( hipFree( swirlDevPtr ) ); 
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Simple OpenGL CUDA");
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// load bitmap	
	Bitmap bmp = Bitmap("who-is-that.bmp");
	if (bmp.isValid())
	{		
		for (int i = 0 ; i < DIM*DIM ; i++) {
			sourceColors[i] = bmp.getR(i/DIM, i%DIM) / 255.0f;
		}
	}

	
	CUDA_SAFE_CALL( hipMalloc((void**)&sourceDevPtr, DIM * DIM )) ;
	CUDA_SAFE_CALL( hipMalloc((void**)&swirlDevPtr, DIM * DIM )) ;
	CUDA_SAFE_CALL( hipMemcpy(sourceDevPtr, sourceColors, DIM * DIM , hipMemcpyHostToDevice) );	
	glutMainLoop();

	cleanup();
}
