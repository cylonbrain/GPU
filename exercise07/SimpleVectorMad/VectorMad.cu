#include "hip/hip_runtime.h"

// Includes
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;

// Functions
void Cleanup(void);
void RandomInit(float*, int);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C)
{
    int i = threadIdx.x;    
    C[i] = A[i] + B[i];	
}

// Host code
int main(int argc, char** argv)
{
    printf("Simple vector addition\n");
    int N = 256;
    size_t size = N * sizeof(float);    

    // Allocate input vectors h_A, h_B and h_C in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();
	
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);	
	
    // Allocate vectors in device memory
    CUDA_SAFE_CALL( hipMalloc((void**)&d_A, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_B, size) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_C, size) );		

    // Copy vectors from host memory to device memory
    CUDA_SAFE_CALL( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );	

	// Invoke kernel
	VecAdd<<<1, N>>>(d_A, d_B, d_C);
	
#ifdef _DEBUG
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
#endif


	// Copy result from device memory to host memory
    // h_C contains the result in host memory
    CUDA_SAFE_CALL( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );

    // Verify result
	// TODO: Print out E and verify the result.
    int i = 0;
    for (i = 0; i < N; ++i) 
	{
        float sum = h_A[i] + h_B[i];
		printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");
    
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

	// TODO: Free device memory of D and E.	

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);

	// TODO: Free host memory of D and E.	
        
    CUDA_SAFE_CALL( hipDeviceReset() );
        
    printf("\nPress ENTER to exit...\n");
    fflush( stdout);
    fflush( stderr);
    getchar();    

    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
